#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdio>

int main(int argc, char *argv[]) {
    int N = atoi(argv[1]);

    float *hA = (float*) malloc(N * N * sizeof(float));
    float *hB = (float*) malloc(N * N * sizeof(float));
    float *hC = (float*) malloc(N * N * sizeof(float));
    
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            hA[i*N+j] = i*N+j + 1;
            if (i == j) hB[i*N+j] = 1;
        }
    }

    float *dA, *dB, *dC;
    hipMalloc(&dA, N * N * sizeof(float));
    hipMalloc(&dB, N * N * sizeof(float));
    hipMalloc(&dC, N * N * sizeof(float));
    
    hipMemcpy(dA, hA, N * N * sizeof(float), hipMemcpyHostToDevice);    
    hipMemcpy(dB, hB, N * N * sizeof(float), hipMemcpyHostToDevice);    

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1, beta = 0;
    hipblasSgemm(
        handle, 
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        N, N, N,
        &alpha,
        dA, N,
        dB, N, 
        &beta, 
        dC, N); 

    hipMemcpy(hC, dC, N * N * sizeof(float), hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            assert(hC[i*N+j] == i*N+j + 1);
        }
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(hA);
    free(hB);
    free(hC);
}
