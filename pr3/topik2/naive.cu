#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void multiply(float *a, float *b, float *c, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; 
    int j = blockIdx.x * blockDim.x + threadIdx.x;
  
    float sum = 0;
    for (int k = 0; k < N; k++) {
        sum = sum + a[i*N + k] * b[k*N + j];
    }

    c[i*N + j] = sum;
}

int main(int argc, char *argv[]) {
    int N = atoi(argv[1]);

    float *hA = (float*) malloc(N * N * sizeof(float));
    float *hB = (float*) malloc(N * N * sizeof(float));
    float *hC = (float*) malloc(N * N * sizeof(float));
    
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            hA[i*N+j] = i*N+j + 1;

            if (i == j) hB[i*N+j] = 1;
        }
    }

    float *dA, *dB, *dC;
    hipMalloc(&dA, N * N * sizeof(float));
    hipMalloc(&dB, N * N * sizeof(float));
    hipMalloc(&dC, N * N * sizeof(float));
    
    hipMemcpy(dA, hA, N * N * sizeof(float), hipMemcpyHostToDevice);    
    hipMemcpy(dB, hB, N * N * sizeof(float), hipMemcpyHostToDevice);    

    dim3 dimBlock(N, N);
    multiply<<<dimBlock, 1>>>(dA, dB, dC, N);
    hipMemcpy(hC, dC, N * N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            assert(abs(hC[i*N+j] - (i*N+j + 1)) < 1e-9);
        }
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(hA);
    free(hB);
    free(hC);
}
