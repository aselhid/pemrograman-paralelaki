#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdio>

int main(int argc, char *argv[]) {
    int N = atoi(argv[1]);

    float *hA = (float*) malloc(N * N * sizeof(float));
    float *hB = (float*) malloc(N * N * sizeof(float));
    float *hC = (float*) malloc(N * N * sizeof(float));
    
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            hA[i*N+j] = 1;
            hB[i*N+j] = i == j ? 1 : 0;
        }
    }

    float *dA, *dB, *dC;
    hipMalloc(&dA, N * N * sizeof(float));
    hipMalloc(&dB, N * N * sizeof(float));
    hipMalloc(&dC, N * N * sizeof(float));
    
    hipMemcpy(dA, hA, N * N * sizeof(float), hipMemcpyHostToDevice);    
    hipMemcpy(dB, hB, N * N * sizeof(float), hipMemcpyHostToDevice);    

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    const float alpha = 1, beta = 0;
    hipblasGemmEx(
	handle,
	HIPBLAS_OP_N,
	HIPBLAS_OP_N,
	N, N, N,
	&alpha,
	dA, HIP_R_32F, N,
	dB, HIP_R_32F, N,
	&beta,
	dC, HIP_R_32F, N,
	HIP_R_32F,
	CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    hipMemcpy(hC, dC, N * N * sizeof(float), hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            assert(hC[i*N+j] == 1);
        }
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(hA);
    free(hB);
    free(hC);
}

